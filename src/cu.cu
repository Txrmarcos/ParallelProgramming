#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <cstdlib> // Para rand() e srand()
#include <ctime>   // Para time()

// Kernel para somar elementos do vetor
__global__ void sumReduce(float *input, float *result, int n) {
    extern __shared__ float shared_data[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Carrega os dados do vetor global para memória compartilhada
    shared_data[tid] = (idx < n) ? input[idx] : 0.0f;
    __syncthreads();

    // Redução em memória compartilhada
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // O resultado da soma é armazenado na primeira posição
    if (tid == 0) {
        atomicAdd(result, shared_data[0]);
    }
}

void part(int n) {
    const int N = n; // Tamanho do vetor (1 milhão de elementos)
    const int BLOCK_SIZE = 256;

    // Inicializar o gerador de números aleatórios
    srand(static_cast<unsigned int>(time(0)));

    // Alocar e inicializar vetor no host (CPU)
    float *h_input = new float[N];
    for (int i = 0; i < N; i++) {
        h_input[i] = static_cast<float>(rand() % 1000001); // Gera valores aleatórios entre 0 e 1000000
    }
    float h_result = 0.0f;

    // Alocar memória no dispositivo (GPU)
    float *d_input, *d_result;
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    // Copiar dados do host para o dispositivo
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Configurar grid e blocos
    int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Medir tempo de execução
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Executar o kernel
    sumReduce<<<grid_size, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_input, d_result, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcular tempo de execução
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copiar o resultado de volta para o host
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Imprimir o resultado no formato decimal
    std::cout << std::fixed << std::setprecision(6);
    std::cout << "Sum: " << h_result << std::endl;
    std::cout << "Tempo de execução: " << milliseconds << " ms" << std::endl;

    // Liberar memória
    delete[] h_input;
    hipFree(d_input);
    hipFree(d_result);
}

int main(){
	part(10000);
	part(100000);
	part(1000000);
	return 0;

}

